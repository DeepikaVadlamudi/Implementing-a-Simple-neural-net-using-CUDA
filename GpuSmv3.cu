
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include "string.h"
#include<time.h>
#include<float.h>

__constant__ int PIC[28*28];

__global__ void conv1(int *filterd, int *resultd){

  int xsize = 28;
  int filterdim = 5;

  __shared__ int fil[25];
  int i,j,l;
  int sum, offset;
  i = threadIdx.y;
  j = threadIdx.x;
  l = blockIdx.x;
  offset = l*25;
  sum =0;
  if(i<filterdim && j<filterdim){
    fil[i*filterdim+j] = filterd[offset + i*filterdim+j];
    // printf("offset: %d, \t fil[%d][%d]:%d\n",offset,i,j,fil[i*filterdim+j]);
  }
  __syncthreads();
  if(i<(xsize -filterdim +1)&& j<(xsize -filterdim +1)){
    sum = fil[0]*PIC[ xsize * (i) + j ] + fil[1]*PIC[ xsize*(i) + (j+1) ]
      + fil[2]*PIC[ xsize * (i)+(j+2)] + fil[3]*PIC[xsize * (i)+(j+3)]
      + fil[4]*PIC[ xsize * (i)+(j+4)]+ fil[5]*PIC[ xsize*(i+1)+(j) ]
      + fil[6]*PIC[ xsize * (i+1) + (j+1) ] + fil[7]*PIC[ xsize*(i+1) + (j+2) ] +
      fil[8]*PIC[ xsize*(i+1) + (j+3) ] + fil[9]*PIC[ xsize*(i+1) + (j+4) ] +
      fil[10]*PIC[ xsize*(i+2) + (j) ]	+ fil[11]*PIC[ xsize * (i+2) + (j+1) ] +
      fil[12]*PIC[ xsize*(i+2) + (j+2)] + fil[13]*PIC[ xsize*(i+2) + (j+3)]
      +fil[14]*PIC[ xsize*(i+2) + (j+4)] + fil[15]*PIC[ xsize*(i+3) + (j)]
      + fil[16]*PIC[ xsize*(i+3) + (j+1)] + fil[17]*PIC[ xsize*(i+3) + (j+2)]
      + fil[18]*PIC[ xsize*(i+3) + (j+3)] + fil[19]*PIC[ xsize*(i+3) + (j+4)]
      + fil[20]*PIC[ xsize*(i+4) + (j)] +fil[21]*PIC[ xsize*(i+3) + (j+1)]
      + fil[22]*PIC[ xsize*(i+4) + (j+2)] + fil[23]*PIC[ xsize*(i+4) + (j+3)]
      + fil[24]*PIC[ xsize*(i+4) + (j+4)];

      resultd[l*(xsize -filterdim +1)*(xsize -filterdim +1) + i*(xsize - filterdim +1)+j] = sum;
      // printf("offset2 : %d \t resultgpu[%d][%d]=%d\n",offset,l,i*(xsize - filterdim +1)+j,resultd[l*(xsize -filterdim +1)*(xsize -filterdim +1) + i*(xsize - filterdim +1)+j]);
  }
}

__global__ void maxpooling(int *maxip1d, int *maxop1d){

  int i,j,l,offset;
  i = threadIdx.y;
  j = threadIdx.x;
  l = blockIdx.x;
  int xsize = 24;
  int filterdim = 5;
  offset = l*xsize*xsize;

  __shared__ int max[576];

  if(i<12 && j<12){
    max[i*2*xsize + j*2] = maxip1d[offset + i*2*xsize + j*2];
    max[i*2*xsize+j*2+1] = maxip1d[offset + i*2*xsize +j*2+1];
    max[i*2*xsize+j*2+24] = maxip1d[offset + i*2*xsize +j*2+24];
    max[i*2*xsize+j*2+25] = maxip1d[offset + i*2*xsize +j*2+25];
    // printf("i: %d,\t j: %d,\t l: %d,\t max1: %d,\t max2: %d,\t max3: %d,\t max4: %d\n",i,j,l,max[i*xsize + j],max[i*xsize+1],max[i*xsize+24],max[i*xsize+25]);

  }

  __syncthreads();

  if(i<12 && j<12){
    int max1, max2;
    if(max[i*xsize + j]>=max[i*xsize + j+1]){
      max1 = max[i*xsize + j];
    }
    else{
      max1 = max[i*xsize + j+1];
    }
    if(max[i*xsize + j+24]>=max[i*xsize + j+25]){
      max2 = max[i*xsize + j+24];
    }
    else{
      max2 = max[i*xsize + j+25];
    }
    if(max1>=max2){
      maxop1d[l*144 + i*12+j]=max1;
      // printf("Max1 : %d\t l: %d \t i: %d\t j: %d\n",max1,l,i,j);
    }
    else{
      maxop1d[l*144 + i*12+j] = max2;
      // printf("Max2 : %d\n",max2);
    }
    // printf("Maxpool1d[%d][%d]:%d\n",l,i*12+j,maxop1d[l*144 + i*12+j]);
  }
}

__global__ void conv2(int *cip2d, int *filter2d, int *cop2d){
  int i,j,l,sum;
  i = threadIdx.y;
  j = threadIdx.x;
  l = blockIdx.x;
  int lstar;
  lstar = l*800;
  sum = 0;
  int k =0;
  int di = 12;
  int disquare = di*di;
  int m;
  if(i<8 && j<8){
    for(m = 0; m<32; m++){
      sum = sum + filter2d[lstar + k]*cip2d[(m*disquare)+ (di*i) + j] + filter2d[lstar + k+1]*cip2d[(m*disquare)+ di*(i) + (j+1)]
        + filter2d[lstar+ k+2]*cip2d[(m*disquare)+ di*(i)+(j+2)] + filter2d[lstar +k+3]*cip2d[(m*disquare)+ di*(i)+(j+3)]
        + filter2d[lstar+k+4]*cip2d[(m*disquare)+ di*(i)+(j+4)]+ filter2d[lstar+ k+5]*cip2d[(m*disquare)+ di*(i+1)+(j)]
        + filter2d[lstar +k+6]*cip2d[(m*disquare)+ di* (i+1) + (j+1) ] + filter2d[lstar+ k+7]*cip2d[(m*disquare)+ di*(i+1)+(j+2)]
        + filter2d[lstar+k+8]*cip2d[(m*disquare)+ di*(i+1) + (j+3) ] + filter2d[lstar +k+9]*cip2d[(m*disquare)+ di*(i+1) +(j+4)]
        + filter2d[lstar+k+10]*cip2d[(m*disquare)+ di*(i+2) +(j)]	+ filter2d[lstar+k+11]*cip2d[(m*disquare)+ di* (i+2) + (j+1)]
        + filter2d[lstar+k+12]*cip2d[(m*disquare)+ di*(i+2) + (j+2)] +filter2d[lstar+k+13]*cip2d[(m*disquare)+ di*(i+2)+(j+3)]
        + filter2d[lstar+k+14]*cip2d[(m*disquare)+ di*(i+2)+(j+4)]+filter2d[lstar+k+15]*cip2d[(m*disquare)+ di*(i+3)+(j)]
        + filter2d[lstar+k+16]*cip2d[(m*disquare)+ di*(i+3)+(j+1)]+filter2d[lstar+k+17]*cip2d[(m*disquare)+ di*(i+3)+(j+2)]
        + filter2d[lstar+k+18]*cip2d[(m*disquare)+ di*(i+3)+(j+3)] + filter2d[lstar+k+19]*cip2d[(m*disquare)+di*(i+3)+(j+4)]
        + filter2d[lstar+k+20]*cip2d[(m*disquare)+ di*(i+4)+(j)] +filter2d[lstar+k+21]*cip2d[(m*disquare)+ di*(i+3)+(j+1)]
        + filter2d[lstar +k+22]*cip2d[(m*disquare)+ di*(i+4)+(j+2)] + filter2d[lstar+k+23]*cip2d[(m*disquare)+ di*(i+4)+(j+3)]
        + filter2d[lstar+ k+24]*cip2d[(m*disquare)+ di*(i+4) + (j+4)];

      k+=25;
    }
    cop2d[l*64+i*8+j] = sum;
    // printf("resultdevice[%d][%d]:%d\n",l,i*8+j,cop2d[l*64+i*8+j]);
  }
}

__global__ void maxpool(int *maxip2d, int *maxop2d){

  int i,j,l;
  i = threadIdx.y;
  j = threadIdx.x;
  l = blockIdx.x;
  int offset;
  offset = l*64;
  int xsize = 12;
  __shared__ int max2[64];

  if(i<8 && j<8){
    max2[i*8 + j] = maxop2d[offset + i*8 +j];
  }
  __syncthreads();
  if(i<4 && j<4){
    int a,b,c,d, m1, m2;
    // index = threadIdx.x*2 + threadIdx.y*2*8;
    a = max2[i*16 + j*2];
    b = max2[i*16 + j*2 +1];
    c = max2[i*16 + j*2+8];
    d = max2[i*16 + j*2 + 9];
    if(a>=b){
      m1 = a;
    }
    else{
      m1 = b;
    }
    if(c>=d){
      m2 = c;
    }
    else{
      m2 = d;
    }
    if(m1>=m2){
      maxop2d[l*16 + i*4+j]=m1;
    }
    else{
      maxop2d[l*16 + i*4+j] = m2;
    }
  }
}

__global__ void dense1(int *denseip1d, int *weight1d, int *denseop1d){
  int i;
  i=threadIdx.x;
  int k;
  int length;
  length = 64*4*4;
  for(k=0;k<length;k++){
    denseop1d[i] += weight1d[i*length + k]*denseip1d[k];
  }
}

__global__ void dense2(int *denseip2d, int *weight2d, int *denseop2d){
  int i;
  i = threadIdx.x;
  int k;
  int length;
  length =64;
  for(k=0;k<length;k++){
    denseop2d[i]+=weight2d[i*length + k]*denseip2d[k];
  }
  // printf("denseop2d[%d]:%d\n",i,denseop2d[i]);
}

int main(int argc, char **argv){
  int xsize;
  int filterdim;
  int numfilters;
  int numfilters1;
  int numunits;
  int numunits1;
  xsize = 28;
  filterdim = 5;
  numfilters = 32;
  numfilters1 = 64;
  numunits = 64;
  numunits1 =10;

  /*Numbytes required for initial image*/
  int numbytes = xsize*xsize*sizeof(int);
  /*Numbytes require for the output of first convolution layer*/
  int numbytes2 = (xsize-filterdim+1)*(xsize-filterdim+1)*sizeof(int); //24x24
  /**Numbytes required for output of first maxpool layer**/
  int numbytes3 = ((xsize-filterdim+1)*(xsize-filterdim+1)/4)*sizeof(int); //12x12
  /*Numbytes required for the output of second convolution layer*/
  int numbytes4 = ((xsize-filterdim+1)/2 - filterdim + 1)*((xsize-filterdim+1)/2 - filterdim + 1)*sizeof(int);//8x8
  /*Numbytes required for the output of second maxpool layer*/
  int numbytes5 = (numbytes4/4);//4x4
  /*Numbytes required for the weight matrix for the first dense layer*/
  int numbytes6 = (numunits*numfilters1*numbytes5);//64x64x4x4

  /*Image on host side*/
  /*Ip and op to first conv layer*/
  unsigned int *pic = (unsigned int *)malloc(numbytes);
  int *result;
  int *filter;
  /*op to first maxpool layer*/
  int *maxop1;
  /*op of second conv layer*/
  int *cop2;
  int *filter2;
  /*op to second maxpool layer*/
  int *maxop2;
  /*op of first dense layer*/
  int *denseop1;
  int *weight1;
  /*op of second dense layer*/
  int *denseop2;
  int *weight2;

  /*Device side variables*/
  int *filterd;
  int *resultd;
  /*Ip and op to first maxpool layer*/
  int *maxip1d;
  int *maxop1d;
  /*Ip and op of second conv layer*/
  int *cip2d;
  int *cop2d;
  int *filter2d;
  /*ip and op to second maxpool layer*/
  int *maxip2d;
  int *maxop2d;
  /*ip and op of first dense layer*/
  int *denseip1d;
  int *denseop1d;
  int *weight1d;
  /*ip and op of second dense layer*/
  int *denseip2d;
  int *denseop2d;
  int *weight2d;

  filter = (int *)malloc( numfilters*filterdim*filterdim*sizeof(int));
  result = (int *)malloc(numfilters*numbytes2);
  maxop1 = (int *)malloc(numfilters*numbytes3);
  cop2 = (int *)malloc(numfilters1*numbytes4);
  filter2 = (int *)malloc(numfilters1*numfilters*filterdim*filterdim*sizeof(int));
  maxop2 = (int *)malloc(numfilters1*numbytes5);
  denseop1 = (int *)malloc(numunits*sizeof(int));
  weight1 = (int *)malloc(numbytes6);
  denseop2 = (int *)malloc(numunits1*sizeof(int));
  weight2 = (int *)malloc(numunits*numunits1*sizeof(int));

  hipMalloc(&filterd,  numfilters*filterdim*filterdim*sizeof(int));
  hipMalloc(&resultd, numfilters*numbytes2);
  hipMalloc(&maxip1d, numfilters*numbytes2);
  hipMalloc(&maxop1d, numfilters*numbytes3);
  hipMalloc(&cip2d, numfilters*numbytes3);
  hipMalloc(&cop2d, numfilters1*numbytes4);
  hipMalloc(&filter2d, numfilters1*numfilters*filterdim*filterdim*sizeof(int));
  hipMalloc(&maxip2d, numfilters1*numbytes4);
  hipMalloc(&maxop2d, numfilters1*numbytes5);
  hipMalloc(&denseip1d, numfilters1*4*4*sizeof(int));
  hipMalloc(&denseop1d, numunits*sizeof(int));
  hipMalloc(&weight1d, numbytes6);
  hipMalloc(&denseip2d, numunits*sizeof(int));
  hipMalloc(&denseop2d, numunits1*sizeof(int));
  hipMalloc(&weight2d, numunits*numunits1*sizeof(int));

  /*Initializing the image on host side*/
  /*Should modify to later on read in image*/
  int i,j,k,l,count,dimx;
  for (i=0; i<xsize; i++) {
   for (j=0; j<xsize; j++) {
     pic[i*xsize + j] = 1;
     //printf("pic[%d][%d] : %d\t",i,j,pic[i*xsize + j]);
   }
   //  printf("\n");
  }

  /*Initializing the filter for first conv layer to a value*/
  /*TO DO : Read in filter from a file */
  for(int k=0;k<numfilters;k++){
   for (int i=0; i<filterdim; i++) {
     for (int j=0; j<filterdim; j++){
       filter[k*(filterdim*filterdim) + i*filterdim + j] = 1;
       // printf("filter[%d][%d]: %d\n",k, i*filterdim + j, filter[k*(filterdim*filterdim) + i*filterdim + j]);
     }
   }
 }

 /*Initializing the filter for second conv layer to a value*/
 /*TO DO : Read in filter from a file */
 for(int k=0;k<numfilters1;k++){
   for(int m= 0; m<numfilters;m++){
     for (int i=0; i<filterdim; i++) {
       for (int j=0; j<filterdim; j++){
         filter2[k*(numfilters*filterdim*filterdim)+ m*filterdim*filterdim + i*filterdim + j] = 1;
         // printf("filter2[%d][%d]: %d\t",k, m*filterdim*filterdim+i*filterdim + j, filter2[k*(numfilters*filterdim*filterdim)+ m*filterdim*filterdim + i*filterdim + j]);
       }
     }
   }
   // printf("\n");
 }

 /*Initializing the weight matrix for first dense layer*/
 int length = 64*16;
 for(l=0;l<numunits;l++){
   for(i=0;i<length;i++){
     weight1[l*length + i] = 1;
   }
 }

 /*Initializing the weight matrix for second dense layer*/
 for(l=0;l<numunits1;l++){
   for(i=0;i<numunits;i++){
     weight2[l*numunits + i] = 1;
   }
 }
 /******************Code that has everything to do with  kernels****************/
 hipEvent_t start, stop;
 hipEventCreate(&start);
 hipEventCreate(&stop);

 hipEventRecord(start,0);

 dim3 dimGrid (32);
 dim3 dimBlock (32,32);

 // cudaMemcpy(picd,pic,numbytes, cudaMemcpyHostToDevice);
 // cudaMemcpyToSymbol(FIL, filter, numfilters*filterdim*filterdim*sizeof(int));

 hipMemcpyToSymbol(HIP_SYMBOL(PIC), pic, numbytes);
 hipMemcpy(filterd, filter, numfilters*filterdim*filterdim*sizeof(int), hipMemcpyHostToDevice);

 conv1<<<dimGrid, dimBlock>>>(filterd,resultd);

 hipMemcpy(result,resultd,numfilters*numbytes2,hipMemcpyDeviceToHost);

 dim3 dimBlock1 (16,16);
 hipMemcpy(maxip1d, result,numfilters*numbytes2, hipMemcpyHostToDevice);

 maxpooling<<<dimGrid, dimBlock1>>>(maxip1d, maxop1d);

 hipMemcpy(maxop1, maxop1d, numfilters*numbytes3, hipMemcpyDeviceToHost);

 hipMemcpy(cip2d, maxop1,numfilters*numbytes3,hipMemcpyHostToDevice);
 hipMemcpy(filter2d, filter2,numfilters1*numfilters*filterdim*filterdim*sizeof(int), hipMemcpyHostToDevice);

 dim3 dimGrid2(64);
 dim3 dimBlock2(8,8);

 conv2<<<dimGrid2, dimBlock2>>>(cip2d, filter2d, cop2d);

 hipMemcpy(cop2, cop2d,numfilters1*numbytes4,hipMemcpyDeviceToHost);

 hipMemcpy(maxip2d, cop2,numfilters1*numbytes4,hipMemcpyHostToDevice);

 maxpool<<<dimGrid2, dimBlock2>>>(maxip2d, maxop2d);

 hipMemcpy(maxop2, maxop2d, numfilters1*numbytes5, hipMemcpyDeviceToHost);

 for(k=0;k<64;k++){
   for(i=0;i<4;i++){
     for(j=0;j<4;j++){
       printf("maxpool[%d][%d]:%d\t",k,i*4+j, maxop1[k*16+i*4+j]);
     }
     printf("\n");
   }
   printf("\n\n");
 }

 hipMemcpy(denseip1d, maxop2, numfilters1*numbytes5, hipMemcpyHostToDevice);
 hipMemcpy(weight1d, weight1, numbytes6, hipMemcpyHostToDevice);

 dim3 dimGrid3(1);
 dim3 dimBlock3(64);
 dense1<<<dimGrid3, dimBlock3>>>(denseip1d, weight1d, denseop1d);

 hipMemcpy(denseop1, denseop1d,numunits*sizeof(int),hipMemcpyDeviceToHost);

 dim3 dimGrid4(1);
 dim3 dimBlock4(10);

 hipMemcpy(denseip2d, denseop1,numunits*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(weight2d, weight2, numunits*numunits1*sizeof(int), hipMemcpyHostToDevice);

 dense2<<<dimGrid4, dimBlock4>>>(denseip2d, weight2d, denseop2d);

 hipMemcpy(denseop2, denseop2d, numunits1*sizeof(int), hipMemcpyDeviceToHost);

 hipEventRecord(stop,0);
 hipEventSynchronize(stop);
 float milliseconds;
 hipEventElapsedTime(&milliseconds, start, stop);
 hipEventDestroy(start);
 hipEventDestroy(stop);
 printf("Time taken : %f seconds", milliseconds/1000);
}
