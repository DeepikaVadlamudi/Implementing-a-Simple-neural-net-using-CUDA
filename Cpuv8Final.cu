/*Done with cpu version of convolution which can be scaled to any number of filters of
size 5x5. Maxpooling done with size 2x2. */
/*Implemented 2 layers of conv2d and maxpool using single 1D array */
/*implemented two dense layers*/
/*To do: measure time taken*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include "string.h"
#include<time.h>

int Maxpooling(int a, int b, int c, int d){
	int temp,i;
	temp = a;
	if(b> c && b>d && b> temp){
		temp = b;
	//	printf("temp is b\n");
	}
	if(c>b && c>d && c> temp){
		temp = c;
		//printf("temp is c\n");
	}
	if(d> c && d>b && d> temp){
		temp = d;
	//	printf("temp is d\n");
	}
//	printf("temp is a\n");
	return temp;
}
int main( int argc, char **argv )
{

	int xsize, filterdim, numfilters, numfilters1, numweights, numweights1;
  xsize = 28;
  filterdim =5;
  numfilters=32;
	numfilters1=64;
	numweights = 64;
	numweights1 = 10;
	/******num bytes required for the initial input********/
	int numbytes =  xsize * xsize * sizeof( int );
	/*******num bytes required for input to conv1********/
  int numbytes2 =  (xsize-filterdim + 1) * (xsize - filterdim +1);
	/*********num bytes required for the input to conv2****/
	int numbytes3 = ((xsize-filterdim + 1)/2 -filterdim + 1) * ((xsize-filterdim + 1)/2 -filterdim + 1);
	/************num of bytes required for first weight matrix**************/
	int numbytes4 =  numfilters1*numweights*numbytes3/4;
	/************num of bytes required for second weight matrix************/
	int numbytes5 = numweights*numweights1;

	/*****Original input - pic*************/
  unsigned int *pic = (unsigned int *)malloc(numbytes);
	/*****filter of the first conv layer*****/
  unsigned int filter[numfilters*filterdim*filterdim];
	/*******filter for the second conv layer*******/
	unsigned int filter2[numfilters1*filterdim*filterdim];
	/*********weight matrix for the first dense layer**********/
	unsigned int weight1[numbytes4];
	/*********weight matrix for the second dense layer**********/
	unsigned int weight2[numbytes5];

	int result[numfilters*numbytes2];
	int result2[numfilters1*numbytes3];
	int maxpool[numfilters*(((xsize-filterdim + 1)*(xsize-filterdim + 1))/4)];
	int maxpool2[numfilters1*(numbytes3/4)];
	int dense1[numweights];
	int dense2[numweights1];

	int i, j;
  int count;
  int sum1,k,l;
	int dimx;
	dimx = numfilters1*(numbytes3/4);

	/*************Should read in input**********/
	for (i=0; i<xsize; i++) {
		for (j=0; j<xsize; j++) {
			pic[i*xsize + j] = 1;
    //  printf("pic[%d][%d] : %d\t",i,j,pic[i*xsize + j]);
		}
  //  printf("\n");
	}
	/******should read in filters*********/
  for(int k=0;k<numfilters;k++){
  	for (int i=0; i<filterdim; i++) {
    	for (int j=0; j<filterdim; j++){
        filter[k*(filterdim*filterdim) + i*filterdim + j] = 1;
	//			printf("filter[%d][%d]: %d\n",k, i*filterdim + j, filter[k*(filterdim*filterdim) + i*filterdim + j]);
    	}
  	 }
	}

	for(int k=0;k<numfilters1;k++){
  	for (int i=0; i<filterdim; i++) {
    	for (int j=0; j<filterdim; j++){
        filter2[k*(filterdim*filterdim) + i*filterdim + j] = 1;
	//			printf("filter2[%d][%d]: %d\n",k, i*filterdim + j, filter2[k*(filterdim*filterdim) + i*filterdim + j]);
    	}
  	 }
	}
	/*********First weight matrix**************/
	for(l=0;l<numweights;l++){
		for(i=0;i<dimx;i++){
			weight1[l*dimx+i] = 1;
			printf("element1 : %d\n", l*dimx+i);
		}
	}
	/**********Second weight matrix**************/
	for(l=0;l<numweights1;l++){
		for(i=0;i<numweights;i++){
			weight2[l*numweights+i] = 1;
			printf("element2 : %d\n", l*numweights+i);
		}
	}
	clock_t start, end;
	start  = clock();
	/*****Operations of first convolutional layer******/
	for(l=0; l < numfilters; l++){
	  count = 0;
		for (i = 0;  i < xsize - filterdim +1; i++){
			for (j = 0; j < xsize - filterdim+1; j++){


	      k =0;

	    sum1 =  (filter[l*(filterdim*filterdim) + k])*pic[ xsize * (i) + j ] + (filter[l*(filterdim*filterdim) + k+1])*pic[ xsize*(i) + (j+1) ]
				+ filter[l*(filterdim*filterdim)+ k+2]*pic[ xsize * (i)+(j+2)] + filter[l*(filterdim*filterdim) +k+3]*pic[xsize * (i)+(j+3)]
				+ filter[l*(filterdim*filterdim) +k+4]*pic[ xsize * (i)+(j+4)]+ filter[l*(filterdim*filterdim) + k+5]*pic[ xsize*(i+1)+(j) ]
				+ filter[l*(filterdim*filterdim) +k+6]*pic[ xsize * (i+1) + (j+1) ] + filter[l*(filterdim*filterdim) + k+7]*pic[ xsize*(i+1) + (j+2) ] +
				filter[l*(filterdim*filterdim) +k+8]*pic[ xsize*(i+1) + (j+3) ] + filter[l*(filterdim*filterdim) +k+9]*pic[ xsize*(i+1) + (j+4) ] +
	      filter[l*(filterdim*filterdim) +k+10]*pic[ xsize*(i+2) + (j) ]	+ filter[l*(filterdim*filterdim) +k+11]*pic[ xsize * (i+2) + (j+1) ] +
	      filter[l*(filterdim*filterdim) +k+12]*pic[ xsize*(i+2) + (j+2)] + filter[l*(filterdim*filterdim) +k+13]*pic[ xsize*(i+2) + (j+3)]
				+filter[l*(filterdim*filterdim) +k+14]*pic[ xsize*(i+2) + (j+4)] + filter[l*(filterdim*filterdim) +k+15]*pic[ xsize*(i+3) + (j)]
				+ filter[l*(filterdim*filterdim) +k+16]*pic[ xsize*(i+3) + (j+1)] + filter[l*(filterdim*filterdim) +k+17]*pic[ xsize*(i+3) + (j+2)]
				+ filter[l*(filterdim*filterdim) +k+18]*pic[ xsize*(i+3) + (j+3)] + filter[l*(filterdim*filterdim) +k+19]*pic[ xsize*(i+3) + (j+4)]
				+ filter[l*(filterdim*filterdim) +k+20]*pic[ xsize*(i+4) + (j)] +filter[l*(filterdim*filterdim) +k+21]*pic[ xsize*(i+3) + (j+1)]
				+ filter[l*(filterdim*filterdim) +k+22]*pic[ xsize*(i+4) + (j+2)] + filter[l*(filterdim*filterdim) +k+23]*pic[ xsize*(i+4) + (j+3)]
				+ filter[l*(filterdim*filterdim) + k+24]*pic[ xsize*(i+4) + (j+4)];


	      result[l*numbytes2 +count] = sum1;
	//      printf("result[%d][%d]=%d\t",l,count,result[l*numbytes2 + count]);
	      count+=1;
			}
//	  	printf("\n");
		}
//		printf("\n\n\n");
	}
	/***************Maxpool***************************/
	for(l=0; l<numfilters; l++){
		count =0;
		for(j=0;j<(xsize-filterdim + 1);j+=2){
				for(i=0;i<(xsize-filterdim + 1);i+=2){
					maxpool[l*(numbytes2/4) + count] =
					Maxpooling(result[l*(xsize - filterdim +1)*(xsize - filterdim +1) + j*(xsize-filterdim+1)+i],
				result[l*(xsize - filterdim +1)*(xsize - filterdim +1) + j*(xsize-filterdim+1)+i+1],
				result[l*(xsize - filterdim +1)*(xsize - filterdim +1) + j*(xsize-filterdim+1)+i+(xsize-filterdim + 1)],
			result[l*(xsize - filterdim +1)*(xsize - filterdim +1) + j*(xsize-filterdim+1)+i+(xsize-filterdim + 1)+1]);
		//	printf("Maxpool[%d][%d] : %d \t",l,count ,maxpool[l*(numbytes2/4) + count]);
			count+=1;
				}
	//			printf("\n");
		}
//		printf("\n\n\n");
	}
	/******Operations of second convolutional layer*******/
	int dim = (xsize-filterdim + 1)/2;
	dimx = (xsize-filterdim + 1)/2 -filterdim + 1;
	printf("dim: %d ; dimx: %d\n", dim,dimx);
	for(l=0; l < numfilters1; l++){
		count = 0;
		for (i = 0;  i < dimx; i++){
			for (j = 0; j < dimx; j++){

				k =0;

				sum1 =  (filter2[l*(filterdim*filterdim) + k])*maxpool[ dim * (i) + j ] + (filter2[l*(filterdim*filterdim) + k+1])*maxpool[ dim*(i) + (j+1) ]
				+ filter2[l*(filterdim*filterdim)+ k+2]*maxpool[ dim * (i)+(j+2)] + filter2[l*(filterdim*filterdim) +k+3]*maxpool[dim * (i)+(j+3)]
				+ filter2[l*(filterdim*filterdim) +k+4]*maxpool[ dim * (i)+(j+4)]+ filter2[l*(filterdim*filterdim) + k+5]*maxpool[ dim*(i+1)+(j) ]
				+ filter2[l*(filterdim*filterdim) +k+6]*maxpool[ dim * (i+1) + (j+1) ] + filter2[l*(filterdim*filterdim) + k+7]*maxpool[ dim*(i+1) + (j+2) ] +
				filter2[l*(filterdim*filterdim) +k+8]*maxpool[ dim*(i+1) + (j+3) ] + filter2[l*(filterdim*filterdim) +k+9]*maxpool[ dim*(i+1) + (j+4) ] +
				filter2[l*(filterdim*filterdim) +k+10]*maxpool[ dim*(i+2) + (j) ]	+ filter2[l*(filterdim*filterdim) +k+11]*maxpool[ dim * (i+2) + (j+1) ] +
				filter2[l*(filterdim*filterdim) +k+12]*maxpool[ dim*(i+2) + (j+2)] + filter2[l*(filterdim*filterdim) +k+13]*maxpool[ dim*(i+2) + (j+3)]
				+filter2[l*(filterdim*filterdim) +k+14]*maxpool[ dim*(i+2) + (j+4)] + filter2[l*(filterdim*filterdim) +k+15]*maxpool[ dim*(i+3) + (j)]
				+ filter2[l*(filterdim*filterdim) +k+16]*maxpool[ dim*(i+3) + (j+1)] + filter2[l*(filterdim*filterdim) +k+17]*maxpool[ dim*(i+3) + (j+2)]
				+ filter2[l*(filterdim*filterdim) +k+18]*maxpool[ dim*(i+3) + (j+3)] + filter2[l*(filterdim*filterdim) +k+19]*maxpool[ dim*(i+3) + (j+4)]
				+ filter2[l*(filterdim*filterdim) +k+20]*maxpool[ dim*(i+4) + (j)] +filter2[l*(filterdim*filterdim) +k+21]*maxpool[ dim*(i+3) + (j+1)]
				+ filter2[l*(filterdim*filterdim) +k+22]*maxpool[ dim*(i+4) + (j+2)] + filter2[l*(filterdim*filterdim) +k+23]*maxpool[ dim*(i+4) + (j+3)]
				+ filter2[l*(filterdim*filterdim) + k+24]*maxpool[ dim*(i+4) + (j+4)];


				result2[l*numbytes3 +count] = sum1;
		//		printf("result2[%d][%d]=%d\t",l,count,result2[l*numbytes3 + count]);
				count+=1;
			}
		//	printf("\n");
		}
	//	printf("\n\n\n");
	}
	/******Second Maxpool Layer******/
	dim =((xsize-filterdim + 1)/2 -filterdim + 1)/2;
	printf("dim: %d ;dimx: %d; numbytes3: %d", dim, dimx, numbytes3);
	for(l=0; l<numfilters1; l++){
		count =0;
		for(j=0;j<dimx;j+=2){
				for(i=0;i<dimx;i+=2){
					maxpool2[l*(numbytes3/4) + count] =
					(Maxpooling(result2[l*numbytes3 + j*dimx+i],
					result2[l*numbytes3 + j*dimx+i+1],
					result2[l*numbytes3 + j*dimx+i+dimx],
					result2[l*numbytes3 + j*dimx+i+dimx+1]))/625;

					printf("Maxpool2[%d][%d] : %d \t",l,count ,maxpool2[l*(numbytes3/4) + count]);
					count+=1;
				}
				printf("\n");
		}
		printf("\n\n\n");
	}
	/********************First Dense layer**********************/
	dimx = numfilters1*(numbytes3/4);

	for(l=0;l<numweights;l++){
		for(i=0;i<dimx;i++){
			dense1[l]+= weight1[l*dimx+i]*maxpool2[i];
		}
		printf("dense1[%d]:%d\n",l,dense1[l]);
	}

	/*************Second Dense Layer***************/
	dimx = numweights;
	for(l=0;l<numweights1;l++){
		for(i=0;i<dimx;i++){
			dense2[l]+= weight2[l*dimx+i]*dense1[i];
		}
		printf("dense2[%d]:%d\n",l,dense2[l]);
	}
	end = clock();
	printf("time taken by cpu : %f seconds",((double) (end - start)) );
}
