
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include "string.h"
// #include<time.h>
#include<float.h>

__constant__ int FIL[32*5*5];

__global__ void conv1(unsigned int *picd, int *resultd){
  int i,j,k,l;
  int sum, offset;
  i = threadIdx.y;
  j = threadIdx.x;
  l = blockIdx.x;
  offset = l*25;
  int xsize = 28;
  int filterdim = 5;
  k=0;
  sum =0;
  if(i<(xsize -filterdim +1)&& j<(xsize -filterdim +1)){
    sum = FIL[offset + k]*picd[ xsize * (i) + j ] + FIL[offset+ k+1]*picd[ xsize*(i) + (j+1) ]
      + FIL[offset+ k+2]*picd[ xsize * (i)+(j+2)] + FIL[offset+k+3]*picd[xsize * (i)+(j+3)]
      + FIL[offset+k+4]*picd[ xsize * (i)+(j+4)]+ FIL[offset+ k+5]*picd[ xsize*(i+1)+(j) ]
      + FIL[offset+k+6]*picd[ xsize * (i+1) + (j+1) ] + FIL[offset+ k+7]*picd[ xsize*(i+1) + (j+2) ] +
      FIL[offset+k+8]*picd[ xsize*(i+1) + (j+3) ] + FIL[offset+k+9]*picd[ xsize*(i+1) + (j+4) ] +
      FIL[offset+k+10]*picd[ xsize*(i+2) + (j) ]	+ FIL[offset+k+11]*picd[ xsize * (i+2) + (j+1) ] +
      FIL[offset+k+12]*picd[ xsize*(i+2) + (j+2)] + FIL[offset+k+13]*picd[ xsize*(i+2) + (j+3)]
      +FIL[offset+k+14]*picd[ xsize*(i+2) + (j+4)] + FIL[offset +k+15]*picd[ xsize*(i+3) + (j)]
      + FIL[offset+k+16]*picd[ xsize*(i+3) + (j+1)] + FIL[offset+k+17]*picd[ xsize*(i+3) + (j+2)]
      + FIL[offset+k+18]*picd[ xsize*(i+3) + (j+3)] + FIL[offset+k+19]*picd[ xsize*(i+3) + (j+4)]
      + FIL[offset+k+20]*picd[ xsize*(i+4) + (j)] +FIL[offset+k+21]*picd[ xsize*(i+3) + (j+1)]
      + FIL[offset+k+22]*picd[ xsize*(i+4) + (j+2)] + FIL[offset+k+23]*picd[ xsize*(i+4) + (j+3)]
      + FIL[offset+ k+24]*picd[ xsize*(i+4) + (j+4)];

      resultd[l*(xsize -filterdim +1)*(xsize -filterdim +1) + i*(xsize - filterdim +1)+j] = sum;
      //printf("resultgpu[%d][%d]=%d\n",l,i*(xsize - filterdim +1)+j,resulth[l*(xsize -filterdim +1)*(xsize -filterdim +1) + i*(xsize - filterdim +1)+j]);
  }
}

__global__ void maxpooling(int *maxip1d, int *maxop1d){

  int i,j,l;
  i = threadIdx.y;
  j = threadIdx.x;
  l = blockIdx.x;
  int xsize = 28;
  int filterdim = 5;
  if(i<((xsize-filterdim+1)/2)&&(j<((xsize-filterdim+1)/2))){
    int a,b,c,d,index, max1, max2;
    index = l*((xsize -filterdim +1)*(xsize -filterdim +1))+ threadIdx.x*2 + threadIdx.y*2*(xsize -filterdim +1);
    a = maxip1d[index];
    b = maxip1d[index +1];
    c = maxip1d[index+(xsize-filterdim+1)];
    d = maxip1d[index + (xsize-filterdim+2)];
    if(a>b){
      max1 = a;
    }
    else{
      max1 = b;
    }
    if(c>d){
      max2 = c;
    }
    else{
      max2 = d;
    }
    if(max1>max2){
      maxop1d[l*(xsize -filterdim +1)*(xsize -filterdim +1)/4 + i*(xsize - filterdim +1)/2+j]=max1;
    }
    else{
      maxop1d[l*(xsize -filterdim +1)*(xsize -filterdim +1)/4 + i*(xsize - filterdim +1)/2+j] = max2;
    }
  }
}

__global__ void conv2(int *cip2d, int *filter2d, int *cop2d){
  int i,j,l,sum;
  i = threadIdx.y;
  j = threadIdx.x;
  l = blockIdx.x;
  int lstar;
  lstar = l*800;
  sum = 0;
  int k =0;
  int di = 12;
  int disquare = di*di;
  int m;
  if(i<8 && j<8){
    for(m = 0; m<32; m++){
      sum = sum + filter2d[lstar + k]*cip2d[(m*disquare)+ (di*i) + j] + filter2d[lstar + k+1]*cip2d[(m*disquare)+ di*(i) + (j+1)]
        + filter2d[lstar+ k+2]*cip2d[(m*disquare)+ di*(i)+(j+2)] + filter2d[lstar +k+3]*cip2d[(m*disquare)+ di*(i)+(j+3)]
        + filter2d[lstar+k+4]*cip2d[(m*disquare)+ di*(i)+(j+4)]+ filter2d[lstar+ k+5]*cip2d[(m*disquare)+ di*(i+1)+(j)]
        + filter2d[lstar +k+6]*cip2d[(m*disquare)+ di* (i+1) + (j+1) ] + filter2d[lstar+ k+7]*cip2d[(m*disquare)+ di*(i+1)+(j+2)]
        + filter2d[lstar+k+8]*cip2d[(m*disquare)+ di*(i+1) + (j+3) ] + filter2d[lstar +k+9]*cip2d[(m*disquare)+ di*(i+1) +(j+4)]
        + filter2d[lstar+k+10]*cip2d[(m*disquare)+ di*(i+2) +(j)]	+ filter2d[lstar+k+11]*cip2d[(m*disquare)+ di* (i+2) + (j+1)]
        + filter2d[lstar+k+12]*cip2d[(m*disquare)+ di*(i+2) + (j+2)] +filter2d[lstar+k+13]*cip2d[(m*disquare)+ di*(i+2)+(j+3)]
        + filter2d[lstar+k+14]*cip2d[(m*disquare)+ di*(i+2)+(j+4)]+filter2d[lstar+k+15]*cip2d[(m*disquare)+ di*(i+3)+(j)]
        + filter2d[lstar+k+16]*cip2d[(m*disquare)+ di*(i+3)+(j+1)]+filter2d[lstar+k+17]*cip2d[(m*disquare)+ di*(i+3)+(j+2)]
        + filter2d[lstar+k+18]*cip2d[(m*disquare)+ di*(i+3)+(j+3)] + filter2d[lstar+k+19]*cip2d[(m*disquare)+di*(i+3)+(j+4)]
        + filter2d[lstar+k+20]*cip2d[(m*disquare)+ di*(i+4)+(j)] +filter2d[lstar+k+21]*cip2d[(m*disquare)+ di*(i+3)+(j+1)]
        + filter2d[lstar +k+22]*cip2d[(m*disquare)+ di*(i+4)+(j+2)] + filter2d[lstar+k+23]*cip2d[(m*disquare)+ di*(i+4)+(j+3)]
        + filter2d[lstar+ k+24]*cip2d[(m*disquare)+ di*(i+4) + (j+4)];

      k+=25;
    }
    cop2d[l*64+i*8+j] = sum;
    // printf("resultdevice[%d][%d]:%d\n",l,i*8+j,cop2d[l*64+i*8+j]);
  }
}

__global__ void maxpool(int *maxip2d, int *maxop2d){

  int i,j,l;
  i = threadIdx.y;
  j = threadIdx.x;
  l = blockIdx.x;
  int xsize = 12;
  int filterdim = 5;
  if(i<((xsize-filterdim+1)/2)&&(j<((xsize-filterdim+1)/2))){
    int a,b,c,d,index, max1, max2;
    index = l*((xsize -filterdim +1)*(xsize -filterdim +1))+ threadIdx.x*2 + threadIdx.y*2*(xsize -filterdim +1);
    a = maxip2d[index];
    b = maxip2d[index +1];
    c = maxip2d[index+(xsize-filterdim+1)];
    d = maxip2d[index + (xsize-filterdim+2)];
    if(a>b){
      max1 = a;
    }
    else{
      max1 = b;
    }
    if(c>d){
      max2 = c;
    }
    else{
      max2 = d;
    }
    if(max1>max2){
      maxop2d[l*(xsize -filterdim +1)*(xsize -filterdim +1)/4 + i*(xsize - filterdim +1)/2+j]=max1;
    }
    else{
      maxop2d[l*(xsize -filterdim +1)*(xsize -filterdim +1)/4 + i*(xsize - filterdim +1)/2+j] = max2;
    }
  }
}

int main(int argc, char **argv){
  int xsize;
  int filterdim;
  int numfilters;
  int numfilters1;
  xsize = 28;
  filterdim = 5;
  numfilters = 32;
  numfilters1 = 64;

  /*Numbytes required for initial image*/
  int numbytes = xsize*xsize*sizeof(int);
  /*Numbytes require for the output of first convolution layer*/
  int numbytes2 = (xsize-filterdim+1)*(xsize-filterdim+1)*sizeof(int); //24x24
  /**Numbytes required for output of first maxpool layer**/
  int numbytes3 = ((xsize-filterdim+1)*(xsize-filterdim+1)/4)*sizeof(int); //12x12
  /*Numbytes required for the output of second convolution layer*/
  int numbytes4 = ((xsize-filterdim+1)/2 - filterdim + 1)*((xsize-filterdim+1)/2 - filterdim + 1)*sizeof(int);//8x8
  /*Numbytes required for the output of second maxpool layer*/
  int numbytes5 = (numbytes4/4)*sizeof(int);//4x4

  /*Image on host side*/
  /*Ip and op to first conv layer*/
  unsigned int *pic = (unsigned int *)malloc(numbytes);
  int *result;
  int filter[numfilters*filterdim*filterdim];
  /*Ip and op to first maxpool layer*/
  int *maxip1;
  int *maxop1;
  /*Ip and op of second conv layer*/
  int *cip2;
  int *cop2;
  int *filter2;
  /*ip and op to second maxpool layer*/
  int *maxip2;
  int *maxop2;

  /*Device side variables*/
  unsigned int *picd;
  int *resultd;
  /*Ip and op to first maxpool layer*/
  int *maxip1d;
  int *maxop1d;
  /*Ip and op of second conv layer*/
  int *cip2d;
  int *cop2d;
  int *filter2d;
  /*ip and op to second maxpool layer*/
  int *maxip2d;
  int *maxop2d;

  result = (int *)malloc(numfilters*numbytes2);
  maxip1 = (int *)malloc(numfilters*numbytes2);
  maxop1 = (int *)malloc(numfilters*numbytes3);
  cip2 = (int *)malloc(numfilters*numbytes3);
  cop2 = (int *)malloc(numfilters1*numbytes4);
  filter2 = (int *)malloc(numfilters1*numfilters*filterdim*filterdim*sizeof(int));
  maxip2 = (int *)malloc(numfilters1*numbytes4);
  maxop2 = (int *)malloc(numfilters1*numbytes5);

  hipMalloc(&picd, numbytes);
  hipMalloc(&resultd, numfilters*numbytes2);
  hipMalloc(&maxip1d, numfilters*numbytes2);
  hipMalloc(&maxop1d, numfilters*numbytes3);
  hipMalloc(&cip2d, numfilters*numbytes3);
  hipMalloc(&cop2d, numfilters1*numbytes4);
  hipMalloc(&filter2d, numfilters1*numfilters*filterdim*filterdim*sizeof(int));
  hipMalloc(&maxip2d, numfilters1*numbytes4);
  hipMalloc(&maxop2d, numfilters1*numbytes5);

  /*Initializing the image on host side*/
  /*Should modify to later on read in image*/
  int i,j,k,l,count,dimx;
  for (i=0; i<xsize; i++) {
   for (j=0; j<xsize; j++) {
     pic[i*xsize + j] = 1;
     //printf("pic[%d][%d] : %d\t",i,j,pic[i*xsize + j]);
   }
   //  printf("\n");
  }

  /*Initializing the filter for first conv layer to a value*/
  /*TO DO : Read in filter from a file */
  for(int k=0;k<numfilters;k++){
   for (int i=0; i<filterdim; i++) {
     for (int j=0; j<filterdim; j++){
       filter[k*(filterdim*filterdim) + i*filterdim + j] = 1;
       // printf("filter[%d][%d]: %d\n",k, i*filterdim + j, filter[k*(filterdim*filterdim) + i*filterdim + j]);
     }
   }
 }

 /*Initializing the filter for second conv layer to a value*/
 /*TO DO : Read in filter from a file */
 for(int k=0;k<numfilters1;k++){
   for(int m= 0; m<numfilters;m++){
     for (int i=0; i<filterdim; i++) {
       for (int j=0; j<filterdim; j++){
         filter2[k*(numfilters*filterdim*filterdim)+ m*filterdim*filterdim + i*filterdim + j] = 1;
         // printf("filter2[%d][%d]: %d\t",k, m*filterdim*filterdim+i*filterdim + j, filter2[k*(numfilters*filterdim*filterdim)+ m*filterdim*filterdim + i*filterdim + j]);
       }
     }
   }
   // printf("\n");
 }

 dim3 dimGrid (32);
 dim3 dimBlock (32,32);

 hipMemcpy(picd,pic,numbytes, hipMemcpyHostToDevice);
 hipMemcpyToSymbol(HIP_SYMBOL(FIL), filter, numfilters*filterdim*filterdim*sizeof(int));

 conv1<<<dimGrid, dimBlock>>>(picd,resultd);

 hipMemcpy(result,resultd,numfilters*numbytes2,hipMemcpyDeviceToHost);

 dim3 dimBlock1 (16,16);
 hipMemcpy(maxip1d, result,numfilters*numbytes2, hipMemcpyHostToDevice);

 maxpooling<<<dimGrid, dimBlock1>>>(maxip1d, maxop1d);

 hipMemcpy(maxop1, maxop1d, numfilters*numbytes3, hipMemcpyDeviceToHost);

 hipMemcpy(cip2d, maxop1,numfilters*numbytes3,hipMemcpyHostToDevice);
 hipMemcpy(filter2d, filter2,numfilters1*numfilters*filterdim*filterdim*sizeof(int), hipMemcpyHostToDevice);

 dim3 dimGrid2(64);
 dim3 dimBlock2(8,8);

 conv2<<<dimGrid2, dimBlock2>>>(cip2d, filter2d, cop2d);

 hipMemcpy(cop2, cop2d,numfilters1*numbytes4,hipMemcpyDeviceToHost);

 hipMemcpy(maxip2d, cop2,numfilters1*numbytes4,hipMemcpyHostToDevice);

 maxpool<<<dimGrid2, dimBlock2>>>(maxip2d, maxop2d);

 hipMemcpy(maxop2, maxop2d, numfilters*numbytes5, hipMemcpyDeviceToHost);

 for(k=0;k<numfilters1;k++){
   for(i=0;i<4;i++){
     for(j=0;j<4;j++){
       printf("maxpool[%d][%d]:%d\t",k,i*4+j, maxop2[k*16+i*4+j]);
     }
     printf("\n");
   }
   printf("\n\n");
 }

}
