/*****Implemented first layer of convolution using global memory*******/
/**Implemented First Maxpool Layer**/
/**Measuring time**/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include "string.h"
// #include<time.h>
#include<float.h>

__constant__ int FIL[32*5*5];

__global__ void conv1(unsigned int *pich, int *resulth, int xsize, int numfilters, int filterdim){
  int i,j,k,l;
  int sum;
  int height;
  i = threadIdx.y;
  j = threadIdx.x;
  l = blockIdx.x;
  k=0;
  sum =0;
  // height = blockIdx.x*(xsize -filterdim +1)*(xsize -filterdim +1);
  if(i<(xsize -filterdim +1)&& j<(xsize -filterdim +1)){
    sum = (FIL[l*(filterdim*filterdim) + k])*pich[ xsize * (i) + j ] + (FIL[l*(filterdim*filterdim) + k+1])*pich[ xsize*(i) + (j+1) ]
      + FIL[l*(filterdim*filterdim)+ k+2]*pich[ xsize * (i)+(j+2)] + FIL[l*(filterdim*filterdim) +k+3]*pich[xsize * (i)+(j+3)]
      + FIL[l*(filterdim*filterdim) +k+4]*pich[ xsize * (i)+(j+4)]+ FIL[l*(filterdim*filterdim) + k+5]*pich[ xsize*(i+1)+(j) ]
      + FIL[l*(filterdim*filterdim) +k+6]*pich[ xsize * (i+1) + (j+1) ] + FIL[l*(filterdim*filterdim) + k+7]*pich[ xsize*(i+1) + (j+2) ] +
      FIL[l*(filterdim*filterdim) +k+8]*pich[ xsize*(i+1) + (j+3) ] + FIL[l*(filterdim*filterdim) +k+9]*pich[ xsize*(i+1) + (j+4) ] +
      FIL[l*(filterdim*filterdim) +k+10]*pich[ xsize*(i+2) + (j) ]	+ FIL[l*(filterdim*filterdim) +k+11]*pich[ xsize * (i+2) + (j+1) ] +
      FIL[l*(filterdim*filterdim) +k+12]*pich[ xsize*(i+2) + (j+2)] + FIL[l*(filterdim*filterdim) +k+13]*pich[ xsize*(i+2) + (j+3)]
      +FIL[l*(filterdim*filterdim) +k+14]*pich[ xsize*(i+2) + (j+4)] + FIL[l*(filterdim*filterdim) +k+15]*pich[ xsize*(i+3) + (j)]
      + FIL[l*(filterdim*filterdim) +k+16]*pich[ xsize*(i+3) + (j+1)] + FIL[l*(filterdim*filterdim) +k+17]*pich[ xsize*(i+3) + (j+2)]
      + FIL[l*(filterdim*filterdim) +k+18]*pich[ xsize*(i+3) + (j+3)] + FIL[l*(filterdim*filterdim) +k+19]*pich[ xsize*(i+3) + (j+4)]
      + FIL[l*(filterdim*filterdim) +k+20]*pich[ xsize*(i+4) + (j)] +FIL[l*(filterdim*filterdim) +k+21]*pich[ xsize*(i+3) + (j+1)]
      + FIL[l*(filterdim*filterdim) +k+22]*pich[ xsize*(i+4) + (j+2)] + FIL[l*(filterdim*filterdim) +k+23]*pich[ xsize*(i+4) + (j+3)]
      + FIL[l*(filterdim*filterdim) + k+24]*pich[ xsize*(i+4) + (j+4)];

      resulth[l*(xsize -filterdim +1)*(xsize -filterdim +1) + i*(xsize - filterdim +1)+j] = sum;
      printf("resultgpu[%d][%d]=%d\n",l,i*(xsize - filterdim +1)+j,resulth[l*(xsize -filterdim +1)*(xsize -filterdim +1) + i*(xsize - filterdim +1)+j]);
  }
}

__global__ void maxpooling(int *resulth, int *maxpoolh, int xsize, int filterdim, int numfilters){

  int i,j,l;
  int temp;
  i = threadIdx.y;
  j = threadIdx.x;
  l = blockIdx.x;
  if(i<((xsize-filterdim+1)/2)&&(j<((xsize-filterdim+1)/2))){
    int a,b,c,d,index, max1, max2;
    index = l*((xsize -filterdim +1)*(xsize -filterdim +1))+ threadIdx.x*2 + threadIdx.y*2*(xsize -filterdim +1);
    a = resulth[index];
    b = resulth[index +1];
    c = resulth[index+(xsize-filterdim+1)];
    d = resulth[index + (xsize-filterdim+2)];
    if(a>b){
      max1 = a;
    }
    else{
      max1 = b;
    }
    if(c>d){
      max2 = c;
    }
    else{
      max2 = d;
    }
    if(max1>max2){
      maxpoolh[l*(xsize -filterdim +1)*(xsize -filterdim +1)/4 + i*(xsize - filterdim +1)/2+j]=max1;
    }
    else{
      maxpoolh[l*(xsize -filterdim +1)*(xsize -filterdim +1)/4 + i*(xsize - filterdim +1)/2+j] = max2;
    }
  }
}
int main( int argc, char **argv )
{

  int xsize;
  int filterdim;
  int numfilters;
  xsize = 28;
  filterdim = 5;
  numfilters =32;

 int numbytes = xsize*xsize*sizeof(int);
 int numbytes2 = (xsize-filterdim+1)*(xsize-filterdim+1)*sizeof(int);
 /**Numbytes required for output of first maxpool layer**/
 int numbytes3 = ((xsize-filterdim+1)*(xsize-filterdim+1)/4)*sizeof(int);

 unsigned int *pic = (unsigned int *)malloc(numbytes);
 unsigned int filter[numfilters*filterdim*filterdim];
 int *result;
 int *maxpool;

 result = (int *)malloc(numfilters*numbytes2);
 maxpool = (int *)malloc(numfilters*numbytes3);

 unsigned int *pich;
 int *resulth;
 int *maxpoolh;

 hipMalloc(&pich, numbytes);
 hipMalloc(&resulth, numfilters*numbytes2);
 hipMalloc(&maxpoolh, numfilters*numbytes3);

 int i,j,k,l,count,dimx;
 for (i=0; i<xsize; i++) {
   for (j=0; j<xsize; j++) {
     pic[i*xsize + j] = 1;
     //printf("pic[%d][%d] : %d\t",i,j,pic[i*xsize + j]);
   }
   //  printf("\n");
 }

 for(int k=0;k<numfilters;k++){
   for (int i=0; i<filterdim; i++) {
     for (int j=0; j<filterdim; j++){
       filter[k*(filterdim*filterdim) + i*filterdim + j] = 1;
			 // printf("filter[%d][%d]: %d\n",k, i*filterdim + j, filter[k*(filterdim*filterdim) + i*filterdim + j]);

     }
   }
 }

 // int blocksize, gridsize;
 dim3 dimGrid (32);
 dim3 dimBlock (32,32);
 // gridsize = numfilters;
 // blocksize = (24,24);
 hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);

 hipMemcpy(pich,pic,numbytes, hipMemcpyHostToDevice);
 hipMemcpyToSymbol(HIP_SYMBOL(FIL), filter, numfilters*filterdim*filterdim*sizeof(int));

 conv1<<<dimGrid, dimBlock>>>(pich, resulth, xsize, numfilters, filterdim);

 hipMemcpy(result,resulth,numfilters*numbytes2,hipMemcpyDeviceToHost);

 dim3 dimBlock1 (16,16);
 hipMemcpy(resulth, result,numfilters*numbytes2, hipMemcpyHostToDevice);

 maxpooling<<<dimGrid, dimBlock1>>>(resulth, maxpoolh, xsize, filterdim, numfilters);

 hipMemcpy(maxpool, maxpoolh, numfilters*numbytes3, hipMemcpyDeviceToHost);


 hipEventRecord(stop,0);
 hipEventSynchronize(stop);
 float time = 0;
 hipEventElapsedTime(&time, start, stop);
 hipEventDestroy(start);
 hipEventDestroy(stop);
 printf("Time taken on GPU: %f ms\n", time);
}
